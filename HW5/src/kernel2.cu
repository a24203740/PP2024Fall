#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define TILE_WIDTH 16

/**************************************************************************************/
/* CUDA MEMCHECK */
/* ref: */
/* https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api */
/**************************************************************************************/
#define gpuErrchk(ans)                                                         \
  {                                                                            \
    gpuAssert((ans), __FILE__, __LINE__);                                      \
  }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = false) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) {
      getchar();
      exit(code);
    }
  }
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,
                             size_t pitch, int maxIterations, int *d_img) {
  // To avoid error caused by the floating number, use the following pseudo code
  int tIDx = blockIdx.x * blockDim.x + threadIdx.x;
  int tIDy = blockIdx.y * blockDim.y + threadIdx.y;
  float x = lowerX + tIDx * stepX;
  float y = lowerY + tIDy * stepY;

  float z_x = x, z_y = y;
  int i;
  for (i = 0; i < maxIterations; ++i) {
    if (z_x * z_x + z_y * z_y > 4.f)
      break;
    float new_x = z_x * z_x - z_y * z_y;
    float new_y = 2.f * z_x * z_y;
    z_x = x + new_x;
    z_y = y + new_y;
  }
  int* target = (int*)((char*)d_img + tIDy * pitch) + tIDx;
  *target = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img,
            int resX, int resY, int maxIterations) {
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;
  size_t imgSize = resX * resY * sizeof(int);
  int *h_img;
  int *d_img;
  size_t pitch = 0;
  gpuErrchk(hipHostAlloc(&h_img, imgSize, hipHostMallocDefault));
  gpuErrchk(hipMallocPitch(&d_img, &pitch, resX * sizeof(int), resY));
  gpuErrchk(hipMemset(d_img, 0, imgSize));

  dim3 dimGrid(resX / TILE_WIDTH, resY / TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
  mandelKernel<<<dimGrid, dimBlock>>>(lowerX, lowerY, stepX, stepY, pitch, maxIterations, d_img);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipMemcpy2D(h_img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost));
  gpuErrchk(hipFree(d_img));

  memcpy(img, h_img, imgSize);

  gpuErrchk(hipHostFree(h_img));
}
